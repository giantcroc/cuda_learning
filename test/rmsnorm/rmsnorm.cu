#include "hip/hip_runtime.h"
#include "rmsnorm.h"
#include "stdio.h"

#define OFFSET(row, id, col) (row*col+id)
#define FLOAT4(data) (reinterpret_cast<float4*>(&(data))[0])

__global__ void rmsnorm_naive(float* in, float* out, const int M, const int N, float eps=1e-5){
    int idx=blockDim.x*blockIdx.x+threadIdx.x;
    if(idx<M){
        float* x=in + idx*N;
        float sum=0;
        for(int i=0;i<N;i++){
            float temp=x[i];
            sum+=temp*temp;
        }
        sum=sqrtf(sum/N+eps);
        float* y=out+idx*N;
        for(int i=0;i<N;i++){
            y[i]=x[i]/sum;
        }
    }
}

int main(void){
    const int M=1,N=4;
    const int Block_size=32, data_size=M*N*sizeof(float);
    float* hin,*hout,*din,*dout, *dhout;
    hin=(float*)malloc(data_size);
    hout=(float*)malloc(data_size);
    dhout=(float*)malloc(data_size);
    for(int i=0;i<M*N;i++){
        hin[i]=i+1;
    }

    hipMalloc(&din, data_size);
    hipMalloc(&dout,data_size);

    hipMemcpy(din,hin,data_size,hipMemcpyDefault);
    dim3 griddim(M);
    dim3 blockdim(Block_size);

    rmsnorm_naive<<<griddim,blockdim>>>(din,dout,M,N);

    hipMemcpy(dhout, dout, data_size,hipMemcpyDefault);

    for(int i=0;i<M*N;i++){
        printf("%f ",dhout[i]);
    }

    hipFree(din);
    hipFree(dout);
    free(hin);
    free(hout);
    free(dhout);
    return 0;
}